/**
 * File              : cuda_conv.cu
 * Author            : Xianglan Piao <xianglan0502@gmail.com>
 * Date              : 2020.06.16
 * Last Modified Date: 2020.06.16
 * Last Modified By  : Xianglan Piao <xianglan0502@gmail.com>
 * NOTE:             : cuda convolution (conv1d(), conv2d())
 */


#include <hip/hip_runtime.h>
#include <iostream>

#define INPUT_SIZE 8
#define KERNEL_SIZE 5
#define STRIDE 2
#define PADDING 2  // For same input/output size PADDING = KERNEL_SIZE / 2
#define OUTPUT_SIZE ((INPUT_SIZE + 2 * PADDING - KERNEL_SIZE) / STRIDE + 1)

//// constant memory
__constant__ float kernel[KERNEL_SIZE];

//// global memory
__global__ void conv1d_naive(float* input, float* output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float temp = 0.0f;
  for (int k = 0; k < KERNEL_SIZE; k++) {
    int col_offset = col * STRIDE - PADDING + k;
    if (col_offset >= 0 && col_offset < INPUT_SIZE) {
      temp += input[col_offset] * kernel[k];
    }
  }

  output[col] = temp;
}

//// shared memory
#define BLOCK_SIZE 4
__global__ void conv1d_shared(float* input, float* output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float shared_block[BLOCK_SIZE];

  float temp = 0.0f;
  for (int k = 0; k < KERNEL_SIZE; k++) {
    int col_offset = col * STRIDE - PADDING + k;

    shared_block[threadIdx.x] = input[col_offset];
    __syncthreads();

    if (col_offset >= 0 && col_offset < INPUT_SIZE) {
      temp += shared_block[threadIdx.x] * kernel[k];
    }
  }

  output[col] = temp;
}

void initData(float* data, int size, float value) {
  for (int i = 0; i < size; i++) {
    data[i] = i + 1;
  }
}

void print1d(float* data, int size) {
  for (int i = 0; i < size; i++) {
    std::cout << data[i] << ", ";
  }
  std::cout << std::endl;
}

//// constant memory
// __constant__ float kernel2d[KERNEL_SIZE * KERNEL_SIZE];

//// global memory
// __global__ void conv2d_naive(float* input, float* output) {
// int col = blockIdx.x * blockDim.x + threadIdx.x;
// int row = blockIdx.y * blockDim.y + threadIdx.y;

// float temp = 0.0f;
// for (int m = 0; m < KERNEL_SIZE; m++) {
// for (int n = 0; n < KERNEL_SIZE; n++) {
// int col_offset = col * STRIDE - PADDING + m;
// int row_offset = row * STRIDE - PADDING + n;
// if ((col_offset >= 0 && col_offset < INPUT_SIZE) &&
// (row_offset >= 0 && row_offset < INPUT_SIZE)) {
// temp += input[row_offset * INPUT_SIZE + col_offset] *
// kernel2d[m * KERNEL_SIZE + n];
// }
// }

// output[row * OUTPUT_SIZE + col] = temp;
// }
// }

int main(void) {
  dim3 block_dim(0);
  dim3 grid_dim(0);

  float* h_input = NULL;
  float* d_input = NULL;
  float* d_output = NULL;
  float* h_output = NULL;

  //// 1D convolution
  std::cout << "\n--- 1D convolution ---\n" << std::endl;
  std::cout << "input: " << std::endl;
  h_input = (float*)malloc(INPUT_SIZE * sizeof(float));
  initData(h_input, INPUT_SIZE, 1.0);
  print1d(h_input, INPUT_SIZE);
  std::cout << std::endl;

  hipMalloc((void**)&d_input, INPUT_SIZE * sizeof(float));
  hipMemcpy(d_input, h_input, INPUT_SIZE * sizeof(float),
             hipMemcpyHostToDevice);

  float h_kernel[KERNEL_SIZE] = {1, 2, 4, 2, 1};
  std::cout << "kernel: " << std::endl;
  print1d(h_kernel, KERNEL_SIZE);
  std::cout << std::endl;
  hipMemcpyToSymbol(HIP_SYMBOL(kernel), &h_kernel, sizeof(kernel));

  block_dim.x = BLOCK_SIZE;
  grid_dim.x = OUTPUT_SIZE / block_dim.x;

  hipMalloc((void**)&d_output, OUTPUT_SIZE * sizeof(float));
  h_output = (float*)calloc(OUTPUT_SIZE, sizeof(float));

  //// using global memory
  hipMemset(d_output, 0, OUTPUT_SIZE * sizeof(float));
  conv1d_naive<<<grid_dim, block_dim>>>(d_input, d_output);
  hipDeviceSynchronize();

  memset(h_output, 0, OUTPUT_SIZE);
  hipMemcpy(h_output, d_output, OUTPUT_SIZE * sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout << "output: " << std::endl;
  print1d(h_output, OUTPUT_SIZE);
  std::cout << std::endl;

  //// using shared memory
  hipMemset(d_output, 0, OUTPUT_SIZE * sizeof(float));
  conv1d_shared<<<grid_dim, block_dim>>>(d_input, d_output);
  hipDeviceSynchronize();

  memset(h_output, 0, OUTPUT_SIZE);
  hipMemcpy(h_output, d_output, OUTPUT_SIZE * sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout << "output: " << std::endl;
  print1d(h_output, OUTPUT_SIZE);
  std::cout << std::endl;

  hipFree(d_input);
  hipFree(d_output);
  free(h_input);
  free(h_output);

  //// 2D convolution
  // printf("\nprint 2D:\n");
  // h_input = (float*)calloc(INPUT_SIZE * size.y, sizeof(float));
  // initInput(h_input, INPUT_SIZE * size.y);
  // cudaMalloc((void**)&input, INPUT_SIZE * size.y * sizeof(float));
  // cudaMemcpy(input, h_input, INPUT_SIZE * size.y * sizeof(float),
  // cudaMemcpyHostToDevice);

  // block_dim.y = 4;
  // grid_dim.y = size.y / block_dim.y;

  // print2D<<<grid_dim, block_dim>>>(d_input);
  // cudaDeviceSynchronize();

  // cudaMemcpy(h_input, d_input, INPUT_SIZE * size.y * sizeof(float),
  // cudaMemcpyDeviceToHost);
  // cudaFree(d_input);
  // free(h_input);

  //// reset
  hipDeviceReset();
}
